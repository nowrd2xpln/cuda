
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
//#include "benchmark.h"

//Macros
#define min(a, b) ( (a)<(b)? (a): (b) )
#define max(a, b) ( (a)>(b)? (a): (b) )

//Constants
#define MAX_VECTOR_COUNT 5

//Vector structure
typedef struct {
	float e[3];
}Vec3f;

//Global array
Vec3f vecArray[MAX_VECTOR_COUNT];
Vec3f newvecArray[MAX_VECTOR_COUNT];

//forward declarations
__global__ void reduce(Vec3f *input, Vec3f *output);

int 
main(int argc, char** argv){

	vecArray[0].e[0] =   1.0; vecArray[0].e[1] =   2.0; vecArray[0].e[2] =   3.0;
	vecArray[1].e[0] =   4.0; vecArray[1].e[1] =   5.0; vecArray[1].e[2] =   6.0;
	vecArray[2].e[0] =   7.0; vecArray[2].e[1] =   8.0; vecArray[2].e[2] =   9.0;
	vecArray[3].e[0] =  10.0; vecArray[3].e[1] =  11.0; vecArray[3].e[2] =  12.0;
	vecArray[4].e[0] =  13.0; vecArray[4].e[1] =  14.0; vecArray[4].e[2] =  15.0;
	// NOTE:  the data being operated on are Vec3f's and frange from 0 (black) to 10ish for each rgb.  
	//I think they are the intesities.

	//--------------------------------------------------------------------------------
	//allocate device mem
	Vec3f *ddata, *dbuffer;

	hipMalloc( &ddata,     MAX_VECTOR_COUNT * sizeof(Vec3f) );
	hipMalloc( &dbuffer,   MAX_VECTOR_COUNT * sizeof(Vec3f) );   
	hipMemset( dbuffer, 0, MAX_VECTOR_COUNT * sizeof(Vec3f) );    

	hipMemcpy( ddata, vecArray, MAX_VECTOR_COUNT * sizeof(Vec3f), hipMemcpyHostToDevice );

	dim3 gridDim(1,1);
	dim3 blockDim(5,1);

	//Check verArray values going into kernel function
	for (int i = 0 ; i < 5 ; i++){
		for (int j = 0 ; j < 3 ; j ++)
	        	printf("vecArray[%d][%d] = %.3f\n", j,i,vecArray[i].e[j]);
    	}
	printf("\n\n");

	//call the reduction function
	reduce<<< gridDim, blockDim >>> ( ddata, dbuffer );

	//ZERO out newvecArray
    	memset(newvecArray, 0, MAX_VECTOR_COUNT * sizeof(Vec3f));
	hipMemcpy( newvecArray, dbuffer, MAX_VECTOR_COUNT * sizeof(Vec3f), hipMemcpyDeviceToHost );
	//Check to see if copied over to newvecArry

	//Check to see if copied over to newvecArry
	printf("Check to see if copied over to newvecArry\n");
	for (int i = 0 ; i < 5 ; i++){
		for (int j = 0 ; j < 3 ; j ++)
	        	printf("newvecArray[%d][%d] = %.3f\n", j,i,newvecArray[i].e[j]);
    	}

	//free device mem
	hipFree( &ddata );

	//--------------------------------------------------------------------------------
	return 0;
}

__global__ void 
reduce(Vec3f *input, Vec3f *output){
	extern __shared__ Vec3f sdata[];

	// each thread loadsome element from global to shared mem
	unsigned int tid = threadIdx.x;
	unsigned int i   = threadIdx.x + blockIdx.x * blockDim.x;
	sdata[tid] = input[i];
	__syncthreads();

	//perform reduction in shared mem
	for(unsigned int s=1; s < blockDim.x; s *= 2) {
		//int s = 2;
		if(tid % (2*s) == 0){

			sdata[tid].e[0] += sdata[tid + s].e[0];	//summing
			sdata[tid].e[1] += sdata[tid + s].e[1];
			sdata[tid].e[2] += sdata[tid + s].e[2];
/*
			sdata[tid].e[0] = min( sdata[tid].e[0], sdata[tid + s].e[0] );	//min
			sdata[tid].e[1] = min( sdata[tid].e[1], sdata[tid + s].e[1] );
			sdata[tid].e[2] = min( sdata[tid].e[2], sdata[tid + s].e[2] );

			sdata[tid].e[0] = max( sdata[tid].e[0], sdata[tid + s].e[0] );	//max
			sdata[tid].e[1] = max( sdata[tid].e[1], sdata[tid + s].e[1] );
			sdata[tid].e[2] = max( sdata[tid].e[2], sdata[tid + s].e[2] );
*/
		}
		__syncthreads();
	}

	// write result for this block to global mem
	if(tid == 0) output[blockIdx.x] = sdata[0];
}
